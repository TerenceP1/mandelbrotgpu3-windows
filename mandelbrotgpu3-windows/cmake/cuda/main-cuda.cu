#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_texture_types.h>
#include <cuda_surface_types.h>
#include <hip/hip_vector_types.h>
using namespace std;

__global__ void add(int* arr1, int* arr2, int* arr3) {
    arr3[threadIdx.x] = arr1[threadIdx.x] + arr2[threadIdx.x];
}

int main() {
    cout << "Hello CMake CUDA!\nLets add some arrays!\n";
    int arr1[]={143, 3, 990, 25, 160, 474, 558, 355, 928, 748, 970, 864, 207, 51, 35, 286, 966, 747, 867, 757, 319, 458, 365, 554, 777, 982, 831, 862, 348, 368, 892};
    int arr2[]={845, 390, 539, 208, 136, 677, 70, 24, 178, 841, 652, 149, 748, 541, 860, 431, 564, 497, 502, 175, 237, 253, 480, 510, 799, 246, 33, 835, 922, 217, 967};
    int *cArr1, *cArr2, *cArr3;
    hipMalloc(&cArr1, sizeof(arr1));
    hipMalloc(&cArr2, sizeof(arr2));
    hipMalloc(&cArr3, sizeof(arr1));
    hipMemcpy(cArr1, arr1, sizeof(arr1), hipMemcpyHostToDevice);
    hipMemcpy(cArr2, arr2, sizeof(arr2), hipMemcpyHostToDevice);
    add<<<1, 31>>>(cArr1, cArr2, cArr3);
    int* arr3;
    hipMemcpy(cArr3, arr3, sizeof(arr2), hipMemcpyDeviceToHost);
    for (int i = 0;i < 31;i++) {
        cout << arr3[i] << ' ';
    }
    cout << endl;
    return 0;
}